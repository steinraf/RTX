#include "hip/hip_runtime.h"
//
// Created by steinraf on 19/08/22.
//

#include "cuda_helpers.h"
#include "ray.h"

#include <iostream>



namespace cuda_helpers{

    __host__ void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
        if (result) {
            std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                      file << ":" << line << " '" << func << "' \n";
            hipDeviceReset();
            exit(99);
        }
    }




    __global__ void initRng(int width, int height, hiprandState *randState){
        int i, j, pixelIndex;
        if (!initIndices(i, j, pixelIndex, width, height)) return;

        hiprand_init(42, pixelIndex, 0, &randState[pixelIndex]);
    }

    __global__ void initVariables(int width, int height){
        int i, j, pixelIndex;
        if (!initIndices(i, j, pixelIndex, 1, 1)) return;

        Vector3f origin{0, 0, 0};
        Vector3f lookAt(0, 0, -1);
        const float distToFocus = (lookAt - origin).norm();

    }

    __global__ void freeVariables(int width, int height){
        int i, j, pixelIndex;
        if (!initIndices(i, j, pixelIndex, 1, 1)) return;


    }

    __device__ Color getColor(const Ray& r){

        if(hitSphere({0,0,-1}, 0.5, r))
            return {1.0f, 0, 0};

        auto t = 0.5*(r.getDirection().normalized()[1] + 1.f);

        return (1-t)*Vector3f{1.f} + t*Color{0.5f, 0.7f, 1.0f};
    }

    __device__ bool hitSphere(const Vector3f& center, float radius, const Ray&r){
        auto oc = r.getOrigin() - center;
        const float a = r.getDirection().squaredNorm();
        const float b = 2.0f * oc.dot(r.getDirection());
        const float c = oc.squaredNorm() - radius*radius;
        const float disc = b*b - 4*a*c;
        return disc > 0;
    }


    __global__ void render(Vector3f *output, int width, int height, hiprandState *localRandState){
        int i, j, pixelIndex;
        if (!initIndices(i, j, pixelIndex, width, height)) return;

//        const float u = (i + hiprand_uniform(localRandState)) / width;
//        const float v = (j + hiprand_uniform(localRandState)) / height;

        const float u = static_cast<float>(i)/(width-1);
        const float v = static_cast<float>(j)/(height-1);


        const float aspectRatio = static_cast<float>(width)/height;

        const float viewportHeight = 2.0f;
        const float viewportWidth = aspectRatio * viewportHeight;
        const float focalLength = 1.0f;

        Vector3f    origin{0.0f},
                    horizontal{viewportWidth, 0, 0},
                    vertical{0, viewportHeight, 0},
                    lowerLeftCorner = origin-horizontal/2.0f-vertical/2.0f - Vector3f{0, 0, focalLength};

        Ray ray{origin, lowerLeftCorner + u*horizontal + v*vertical - origin};

        auto col = getColor(ray);

        output[pixelIndex] = col;
    }



}
